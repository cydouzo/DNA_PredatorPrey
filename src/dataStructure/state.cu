#include "state.h"
#include <stdio.h>
#include <assert.h>

template <typename T>
__host__ State<T>::State(const int nSp, const int samSizX, const int samSizY, bool isDevice)
    : nSpecies(nSp), sampleSizeX(samSizX), sampleSizeY(samSizY),
      subSampleSizeX(samSizX), subSampleSizeY(samSizY), isDeviceData(isDevice)
{
    if (isDevice)
    {
        hipMalloc(&data, GetSize() * sizeof(T));
    }
    else
    {
        data = new T[GetSize()];
    }
}

template <typename T>
__host__ State<T>::State(State<T> &state, bool isDevice)
    : State(state.nSpecies, state.sampleSizeX, state.sampleSizeY, isDevice)
{
    assert(state.isDeviceData == false);
    if (isDevice)
        hipMemcpy(data, state.GetRawData(), GetSize() * sizeof(T),
                   hipMemcpyHostToDevice);
    else
    {
        throw "Cannot make these kind of copies yet";
    }
}

template <typename T>
__device__ __host__ T &State<T>::operator()(int s, int x, int y)
{
    if (s < 0 || s > nSpecies)
    {
        printf("Invalid species: %i\n", s);
        return tdefVal;
    }
    if (x == -1 || x == sampleSizeX || y == sampleSizeY || y == -1)
        return tdefVal;
    if (x < -1 || x > sampleSizeX || y > sampleSizeY || y < -1)
    {
        printf("Invalid position: %i %i\n", x, y);
        return tdefVal;
    }
    return data[s * sampleSizeX * sampleSizeY + x * sampleSizeY + y];
}
template <typename T>
__device__ __host__ T &State<T>::operator()(int s, int x)
{
    return this->operator()(s, x, 0);
}

template <typename T>
__device__ __host__ T &State<T>::operator()(dim3 position)
{
    return this->operator()(position.x, position.y, position.z);
}

template <typename T>
__device__ __host__ T *State<T>::GetRawData()
{
    return data;
}

template <typename T>
__device__ __host__ int State<T>::GetSize()
{
    return nSpecies * sampleSizeX * sampleSizeY;
}

template <typename T>
__host__ State<T>::~State()
{
    delete[] data;
}

template __device__ __host__ class State<double>;
template __device__ __host__ class State<float>;