#include "hip/hip_runtime.h"
#include <iostream>
#include "predator_prey_systems_cuda.hpp"
#include <assert.h>
#include <constants.hpp>
#include <functions.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ dim3 position()
{
    return dim3(blockIdx.x, threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y);
}
__global__ void differentiate(double* x, double* dxdt, int im, int jm, double t, double dt)
{
    int i = blockIdx.x;
    int j = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y;
    //i = position().x; j = position().y;
    if((j+1)%jm > 1)
        dxdt[j + jm * i] = (i == 0)? devPreyFunction(x[j], x[j + jm], devLaplacien(&x[j]))
                : devPredatorFunction(x[j], x[j+jm], devLaplacien( &x[j+jm]));
    else
        dxdt[j + jm * i] = 0;
}
__global__ void addStep(double* x, double* dxdt,int im, int jm, double t, double dt)
{
    int i = blockIdx.x;
    int j = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y;
    x[j + jm * i] = dxdt[j + jm * i] * dt;
}

prey_predator_iterator::prey_predator_iterator(double *x, int im, int jm, double snapPeriod = 0.0){
    this->im = im;
    this->jm = jm;
    gpuErrchk(  hipMalloc(&this->x, im * jm * sizeof(double)) );
    gpuErrchk(  hipMalloc(&this->dxdt, im * jm * sizeof(double)) );
    gpuErrchk(  hipMemcpy(this->x, x, im * jm * sizeof(double), hipMemcpyHostToDevice) );
    this->snapPeriod = snapPeriod;
    dim3 threadsPerBlock(10, 30);
    dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
   // hipMemcpy(xDevice, x, im*jm*sizeof(double), hipMemcpyHostToDevice);
}
prey_predator_iterator::~prey_predator_iterator(){
};

void prey_predator_iterator::iterate(double t, double dt){
    // dim3 threadsPerBlock(32, 32);
    dim3 threadsPerBlock(10, 30);
    dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
      printer(t,dt,0.0);
    differentiate<<<numBlocks,threadsPerBlock>>>(x, dxdt, im, jm, t, dt);
    addStep<<<numBlocks,threadsPerBlock>>>(x, dxdt, im, jm, t, dt);
    printer(t,dt,1.0);
    printer(t,dt, 9.0); 
    //TODO check why x doesn't change
}

void prey_predator_iterator::printer(double t, double dt, double tp){
    if(t >= tp && t<tp+dt){
        std::cout << std::endl;
        dim3 threadsPerBlock(10, 30);
        dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
        double *xHost = new double[im * jm];
        hipMemcpy(xHost, x, im*jm*sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i<im*jm; i++){
            std::cout << xHost[i] << " ";
        }
        std::cout << std::endl;
    }

}