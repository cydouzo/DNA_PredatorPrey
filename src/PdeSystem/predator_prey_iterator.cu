#include "hip/hip_runtime.h"
#include <iostream>
#include "predator_prey_systems_cuda.hpp"
#include <assert.h>
#include <function.h>
#include <constants.hpp>

__shared__ double *xDevice;
__shared__ double *dxdtDevice;

__device__ dim3 position()
{
    return dim3(blockIdx.x, threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y);
}
__global__ void differentiate(int im, int jm, double t, double dt)
{
    int i = blockIdx.x;
    int j = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y;
    i = position().x; j = position().y;
    if((j+1)%jm > 1)
        dxdtDevice[j + jm * i] = (i == 0)? devPreyFunction(xDevice[j], xDevice[j + jm], devLaplacien(&xDevice[j]))
                : devPredatorFunction(xDevice[j], xDevice[j], devLaplacien( &xDevice[j+jm]));
    else
        dxdtDevice[j + jm * i] = 0;
}
__global__ void addStep(int im, int jm, double t, double dt)
{
    int i = blockIdx.x;
    int j = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.y * blockDim.x * blockDim.y;
    xDevice[j + jm * i] = dxdtDevice[j + jm * i] * dt;
}
__global__ void copyMemoryToDevice(double *x){
    int i = position().x; 
    int j = position().y;
    xDevice[j + i * size2] = x[j + i * size2];
}

__global__ void copyMemoryToHost(double *x){
    int i = position().x; 
    int j = position().y;
    x[j + i * size2] = xDevice[j + i * size2];
}

prey_predator_iterator::prey_predator_iterator(double *x, int im, int jm, double snapPeriod = 0.0){
    this->im = im;
    this->jm = jm;
    this->x = x;
    this->snapPeriod = snapPeriod;
	hipMalloc(&x, size1 * size2 * sizeof(double));
    dim3 threadsPerBlock(10, 30);
    dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
    copyMemoryToDevice<<<numBlocks,threadsPerBlock,600* sizeof(double)>>>(x);
    //hipMemcpy(xDevice, x, im*jm*sizeof(double), hipMemcpyHostToDevice);
}
prey_predator_iterator::~prey_predator_iterator(){
};

void prey_predator_iterator::iterate(double t, double dt){
    // dim3 threadsPerBlock(32, 32);
    dim3 threadsPerBlock(10, 30);
    dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
    differentiate<<<numBlocks,threadsPerBlock>>>(im, jm, t, dt);
    addStep<<<numBlocks,threadsPerBlock>>>(im, jm, t, dt);
    printer(t,dt,1.0);
    printer(t,dt,0.0);
    printer(t,dt, 9.0); 
    //TODO check why x doesn't change
}

void prey_predator_iterator::printer(double t, double dt, double tp){
    if(t >= tp && t<tp+dt){
        dim3 threadsPerBlock(10, 30);
        dim3 numBlocks(2);//, im * jm / (threadsPerBlock.x * threadsPerBlock.y));
        copyMemoryToHost<<<numBlocks,threadsPerBlock,600 * sizeof(double)>>>(x);
        for (int i = 0; i<im*jm; i++){
            std::cout << x[i] << " ";
        }
        std::cout << std::endl;
        std::cout << std::endl;
    }

}